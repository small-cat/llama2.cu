/* Inference for Llama-2 Transformer model in pure C */
/* With CUDA support that draws heavily from https://github.com/ankan-ban/llama2.cu/blob/master/llama2.cu */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <ctype.h>
#include <time.h>
#include <math.h>
#include <string.h>
#include <fcntl.h>
#if defined _WIN32
    #include "win.h"
#else
    #include <unistd.h>
    #include <sys/mman.h>
#endif

#ifdef USE_CUDA
#include <cuda.h>
#include <cuda_runtime.h>
#include <cub/cub.cuh>
#endif

#ifdef USE_CUDA
#define CUCHK(err) cuda_check((err), __FILE__, __LINE__)
inline void cuda_check(cudaError_t error_code, const char *file, int line)
{
    if (error_code != cudaSuccess)
    {
        fprintf(stderr, "CUDA Error %d: %s. In file '%s' on line %d\n", error_code, cudaGetErrorString(error_code), file, line);
        fflush(stderr);
        exit(error_code);
    }
}
#endif


// ----------------------------------------------------------------------------
// Transformer and RunState structs, and related memory management

typedef struct {
    int dim; // transformer dimension
    int hidden_dim; // for ffn layers
    int n_layers; // number of layers
    int n_heads; // number of query heads
    int n_kv_heads; // number of key/value heads (can be < query heads because of multiquery)
    int vocab_size; // vocabulary size, usually 256 (byte-level)
    int seq_len; // max sequence length
} Config;

typedef struct {
    // token embedding table
    float* token_embedding_table;    // (vocab_size, dim)
    // weights for rmsnorms
    float* rms_att_weight; // (layer, dim) rmsnorm weights
    float* rms_ffn_weight; // (layer, dim)
    // weights for matmuls. note dim == n_heads * head_size
    float* wq; // (layer, dim, n_heads * head_size)
    float* wk; // (layer, dim, n_kv_heads * head_size)
    float* wv; // (layer, dim, n_kv_heads * head_size)
    float* wo; // (layer, n_heads * head_size, dim)
    // weights for ffn
    float* w1; // (layer, hidden_dim, dim)
    float* w2; // (layer, dim, hidden_dim)
    float* w3; // (layer, hidden_dim, dim)
    // final rmsnorm
    float* rms_final_weight; // (dim,)
    // (optional) classifier weights for the logits, on the last layer
    float* wcls;
} TransformerWeights;

typedef struct {
    // current wave of activations
    float *x; // activation at current time stamp (dim,)
    float *xb; // same, but inside a residual branch (dim,)
    float *xb2; // an additional buffer just for convenience (dim,)
    float *hb; // buffer for hidden dimension in the ffn (hidden_dim,)
    float *hb2; // buffer for hidden dimension in the ffn (hidden_dim,)
    float *q; // query (dim,)
    float *k; // key (dim,)
    float *v; // value (dim,)
    float *att; // buffer for scores/attention values (n_heads, seq_len)
#ifdef USE_CUDA
    float *logits_gpu; // output logits in GPU
#endif
    float *logits; // output logits in CPU
    // kv cache
    float* key_cache;   // (layer, seq_len, dim)
    float* value_cache; // (layer, seq_len, dim)
} RunState;

#ifdef USE_CUDA
// RunState is stored on GPU
void malloc_run_state(RunState* s, Config* p) {
    // we calloc instead of malloc to keep valgrind happy
    int kv_dim = (p->dim * p->n_kv_heads) / p->n_heads;
    CUCHK(cudaMalloc((void**)&s->x, p->dim * sizeof(float)));
    CUCHK(cudaMalloc((void**)&s->xb, p->dim * sizeof(float)));
    CUCHK(cudaMalloc((void**)&s->xb2, p->dim * sizeof(float)));
    CUCHK(cudaMalloc((void**)&s->hb, p->hidden_dim * sizeof(float)));
    CUCHK(cudaMalloc((void**)&s->hb2, p->hidden_dim * sizeof(float)));
    CUCHK(cudaMalloc((void**)&s->q, p->dim * sizeof(float)));
    CUCHK(cudaMalloc((void**)&s->k, kv_dim * sizeof(float)));
    CUCHK(cudaMalloc((void**)&s->v, kv_dim * sizeof(float)));
    CUCHK(cudaMalloc((void**)&s->att, p->n_heads * p->seq_len * sizeof(float)));
    CUCHK(cudaMalloc((void**)&s->logits_gpu, p->vocab_size * sizeof(float)));
    s->logits = (float *)calloc(p->vocab_size, sizeof(float));
    CUCHK(cudaMalloc((void**)&s->key_cache, p->n_layers * p->seq_len * kv_dim * sizeof(float)));
    CUCHK(cudaMalloc((void**)&s->value_cache, p->n_layers * p->seq_len * kv_dim * sizeof(float)));
    // ensure all mallocs went fine
    if (!s->x || !s->xb || !s->xb2 || !s->hb || !s->hb2 || !s->q
     || !s->k || !s->v || !s->att || !s->logits_gpu || !s->logits || !s->key_cache
     || !s->value_cache) {
        fprintf(stderr, "malloc failed!\n");
        exit(EXIT_FAILURE);
    }
}
#else
void malloc_run_state(RunState* s, Config* p) {
    // we calloc instead of malloc to keep valgrind happy
    int kv_dim = (p->dim * p->n_kv_heads) / p->n_heads;
    s->x = (float *)calloc(p->dim, sizeof(float));
    s->xb = (float *)calloc(p->dim, sizeof(float));
    s->xb2 = (float *)calloc(p->dim, sizeof(float));
    s->hb = (float *)calloc(p->hidden_dim, sizeof(float));
    s->hb2 = (float *)calloc(p->hidden_dim, sizeof(float));
    s->q = (float *)calloc(p->dim, sizeof(float));
    s->k = (float *)calloc(kv_dim, sizeof(float));
    s->v = (float *)calloc(kv_dim, sizeof(float));
    s->att = (float *)calloc(p->n_heads * p->seq_len, sizeof(float));
    s->logits = (float *)calloc(p->vocab_size, sizeof(float));
    s->key_cache = (float *)calloc(p->n_layers * p->seq_len * kv_dim, sizeof(float));
    s->value_cache = (float *)calloc(p->n_layers * p->seq_len * kv_dim, sizeof(float));
    // ensure all mallocs went fine
    if (!s->x || !s->xb || !s->xb2 || !s->hb || !s->hb2 || !s->q
     || !s->k || !s->v || !s->att || !s->logits || !s->key_cache
     || !s->value_cache) {
        fprintf(stderr, "malloc failed!\n");
        exit(EXIT_FAILURE);
    }
}
#endif

#ifdef USE_CUDA
void free_run_state(RunState* s) {
    CUCHK(cudaFree(s->x));
    CUCHK(cudaFree(s->xb));
    CUCHK(cudaFree(s->xb2));
    CUCHK(cudaFree(s->hb));
    CUCHK(cudaFree(s->hb2));
    CUCHK(cudaFree(s->q));
    CUCHK(cudaFree(s->k));
    CUCHK(cudaFree(s->v));
    CUCHK(cudaFree(s->att));
    CUCHK(cudaFree(s->logits_gpu));
    free(s->logits);
    CUCHK(cudaFree(s->key_cache));
    CUCHK(cudaFree(s->value_cache));
}
#else
void free_run_state(RunState* s) {
    free(s->x);
    free(s->xb);
    free(s->xb2);
    free(s->hb);
    free(s->hb2);
    free(s->q);
    free(s->k);
    free(s->v);
    free(s->att);
    free(s->logits);
    free(s->key_cache);
    free(s->value_cache);
}
#endif

// ----------------------------------------------------------------------------
// initialization: read from checkpoint

void checkpoint_init_weights(TransformerWeights *w, Config* p, float* ptr, int shared_weights) {
    int head_size = p->dim / p->n_heads;
    w->token_embedding_table = ptr;
    ptr += p->vocab_size * p->dim;
    w->rms_att_weight = ptr;
    ptr += p->n_layers * p->dim;
    w->wq = ptr;
    ptr += p->n_layers * p->dim * (p->n_heads * head_size);
    w->wk = ptr;
    ptr += p->n_layers * p->dim * (p->n_kv_heads * head_size);
    w->wv = ptr;
    ptr += p->n_layers * p->dim * (p->n_kv_heads * head_size);
    w->wo = ptr;
    ptr += p->n_layers * (p->n_heads * head_size) * p->dim;
    w->rms_ffn_weight = ptr;
    ptr += p->n_layers * p->dim;
    w->w1 = ptr;
    ptr += p->n_layers * p->dim * p->hidden_dim;
    w->w2 = ptr;
    ptr += p->n_layers * p->hidden_dim * p->dim;
    w->w3 = ptr;
    ptr += p->n_layers * p->dim * p->hidden_dim;
    w->rms_final_weight = ptr;
    ptr += p->dim;
    ptr += p->seq_len * head_size / 2; // skip what used to be freq_cis_real (for RoPE)
    ptr += p->seq_len * head_size / 2; // skip what used to be freq_cis_imag (for RoPE)
    w->wcls = shared_weights ? w->token_embedding_table : ptr;
}

void read_checkpoint(char* checkpoint, Config* config, TransformerWeights* weights,
                     int* fd, float** data, ssize_t* file_size) {
    FILE *file = fopen(checkpoint, "rb");
    if (!file) { fprintf(stderr, "Couldn't open file %s\n", checkpoint); exit(EXIT_FAILURE); }
    // read in the config header
    if (fread(config, sizeof(Config), 1, file) != 1) { exit(EXIT_FAILURE); }
    // negative vocab size is hacky way of signaling unshared weights. bit yikes.
    int shared_weights = config->vocab_size > 0 ? 1 : 0;
    config->vocab_size = abs(config->vocab_size);
    // figure out the file size
    fseek(file, 0, SEEK_END); // move file pointer to end of file
    *file_size = ftell(file); // get the file size, in bytes
    fclose(file);
    // memory map the Transformer weights into the data pointer
    *fd = open(checkpoint, O_RDONLY); // open in read only mode
    if (*fd == -1) { fprintf(stderr, "open failed!\n"); exit(EXIT_FAILURE); }
    *data = (float *)mmap(NULL, *file_size, PROT_READ, MAP_PRIVATE, *fd, 0);
    if (*data == MAP_FAILED) { fprintf(stderr, "mmap failed!\n"); exit(EXIT_FAILURE); }
#ifdef USE_CUDA
    // copy mmap data to the gpu first
    float* weights_ptr;
    size_t weights_size = *file_size - sizeof(Config);
    CUCHK(cudaMalloc((void**)&weights_ptr, weights_size)); // FIXME cudaFree
    CUCHK(cudaMemcpy(weights_ptr, *data + sizeof(Config)/sizeof(float), weights_size, cudaMemcpyHostToDevice));
#else
    float* weights_ptr = *data + sizeof(Config)/sizeof(float);
#endif
    checkpoint_init_weights(weights, config, weights_ptr, shared_weights);
}

// ----------------------------------------------------------------------------
// neural net blocks
#ifdef USE_CUDA
// Single block - not enough parallelism for the GPU, but it's just 1% of total time
__global__ void rmsnorm_kernel(float* o, float* x, float* weight, int size, int elementsPerThread) {
    float ss = 0.0f;
    for (int i = 0; i < elementsPerThread; i++) {
        int index = threadIdx.x + i * 1024;
        if (index < size)
            ss += (float)x[index];
    }

    using BlockReduce = cub::BlockReduce<float, 1024>;
    __shared__ typename BlockReduce::TempStorage temp;
    ss = BlockReduce(temp).Sum(ss * ss);

    __shared__ float shared_ss;
    if (threadIdx.x == 0) {
        ss /= size;
        ss += 1e-5f;
        ss = 1.0f / sqrtf(ss);
        shared_ss = ss;
    }
    __syncthreads();
    ss = shared_ss;

    // normalize
    for (int i = 0; i < elementsPerThread; i++) {
        int index = threadIdx.x + i * 1024;
        if (index < size) {
            float val = (float)x[index];
            val *= ss * (float)weight[index];
            o[index] = val;
        }
    }
}

// divide a into ceiling of b parts
int divUp(int a, int b) {
    return (a - 1) / b + 1;
}

void rmsnorm(float* o, float* x, float* weight, int size) {
    int elementsPerThread = divUp(size, 1024);
    rmsnorm_kernel <<<1, 1024 >>> (o, x, weight, size, elementsPerThread);
    }
#else
void rmsnorm(float* o, float* x, float* weight, int size) {
    // calculate sum of squares
    float ss = 0.0f;
    for (int j = 0; j < size; j++) {
        ss += x[j] * x[j];
    }
    ss /= size;
    ss += 1e-5f;
    ss = 1.0f / sqrtf(ss);
    // normalize and scale
    for (int j = 0; j < size; j++) {
        o[j] = weight[j] * (ss * x[j]);
    }
}
#endif

#ifdef USE_CUDA
__device__ void softmax_gpu(float* __restrict__ x, int size) {
    using BlockReduce = cub::BlockReduce<float, 1024>;
    __shared__ typename BlockReduce::TempStorage temp;
    __shared__ float shared_val;

    int tid = threadIdx.x;
    int step = blockDim.x;

    // find max value (for numerical stability)
    float max_val = tid < size ? x[tid] : 0;
    for (int i = tid + step; i < size; i += step)
        if (x[i] > max_val)
            max_val = x[i];

    max_val = BlockReduce(temp).Reduce(max_val, cub::Max());
    if (threadIdx.x == 0)
        shared_val = max_val;
    __syncthreads();
    max_val = shared_val;

    // exp and sum
    float sum = 0.0f;
    for (int i = tid; i < size; i += step) {
        x[i] = expf(x[i] - max_val);
        sum += x[i];
    }

    sum = BlockReduce(temp).Sum(sum);
    if (threadIdx.x == 0)
        shared_val = sum;
    __syncthreads();
    sum = shared_val;

    // normalize
    for (int i = tid; i < size; i += step)
        x[i] /= sum;
}
#endif
void softmax(float* x, int size) {
    // find max value (for numerical stability)
    float max_val = x[0];
    for (int i = 1; i < size; i++) {
        if (x[i] > max_val) {
            max_val = x[i];
        }
    }
    // exp and sum
    float sum = 0.0f;
    for (int i = 0; i < size; i++) {
        x[i] = expf(x[i] - max_val);
        sum += x[i];
    }
    // normalize
    for (int i = 0; i < size; i++) {
        x[i] /= sum;
    }
}

#ifdef USE_CUDA
// one output per warp so that we can parallelize the dot product across the warp
// Note that ~95% of total time is spent here, so optimizing this is important
__global__ void mat_vec_kernel(float* output, float* input, float* weight, int n, int d, int numSerialElements) {
    int index = blockIdx.x * blockDim.y + threadIdx.y;
    if (index >= d)
        return;

    float sum = 0;
    for (int i = 0; i < numSerialElements; i++) {
        int j = i * 32 + threadIdx.x;
        if (j < n)
            sum += (weight[index * n + j]) * (input[j]);
    }

    using WarpReduce = cub::WarpReduce<float>;
    __shared__ typename WarpReduce::TempStorage temp;
    sum = WarpReduce(temp).Sum(sum);

    if (threadIdx.x == 0)
        output[index] = sum;
}

void matmul(float* xout, float* x, float* w, int n, int d) {
    int serialElements = divUp(n, 32);
    dim3 block_dim(32, 4);
    int blocks = divUp(d, 4);
    mat_vec_kernel <<<blocks, block_dim >>> (xout, x, w, n, d, serialElements);
}
#else
void matmul(float* xout, float* x, float* w, int n, int d) {
    // W (d,n) @ x (n,) -> xout (d,)
    // by far the most amount of time is spent inside this little function
    int i;
    #pragma omp parallel for private(i)
    for (i = 0; i < d; i++) {
        float val = 0.0f;
        for (int j = 0; j < n; j++) {
            val += w[i * n + j] * x[j];
        }
        xout[i] = val;
    }
}
#endif

// Additional neural net blocks (brought out from transformer function)
#ifdef USE_CUDA
__global__ void RoPe_rotation_kernel(int pos, float *sq, float *sk, int kv_dim, int head_size) {
    int i = threadIdx.x * 2;
    int head_dim = i % head_size;
    float freq = 1.0f / powf(10000.0f, head_dim / (float)head_size);
    float val = pos * freq;
    float fcr = cosf(val);
    float fci = sinf(val);
    int rotn = i < kv_dim ? 2 : 1; // how many vectors? 2 = q & k, 1 = q only
    for (int v = 0; v < rotn; v++) {
        float* vec = v == 0 ? sq : sk; // the vector to rotate (query or key)
        float v0 = vec[i];
        float v1 = vec[i+1];
        vec[i]   = v0 * fcr - v1 * fci;
        vec[i+1] = v0 * fci + v1 * fcr;
    }
}
void RoPe_rotation(int pos, RunState* s, int dim, int kv_dim, int head_size) {
    RoPe_rotation_kernel <<<1, dim/2 >>> (pos, s->q, s->k, kv_dim, head_size);
}
#else
void RoPe_rotation(int pos, RunState* s, int dim, int kv_dim, int head_size) { //s->q, s->k, freq_cis_real_row, freq_cis_imag_row, p->n_heads, head_size) {
    for (int i = 0; i < dim; i+=2) {
        int head_dim = i % head_size;
        float freq = 1.0f / powf(10000.0f, head_dim / (float)head_size);
        float val = pos * freq;
        float fcr = cosf(val);
        float fci = sinf(val);
        int rotn = i < kv_dim ? 2 : 1; // how many vectors? 2 = q & k, 1 = q only
        for (int v = 0; v < rotn; v++) {
            float* vec = v == 0 ? s->q : s->k; // the vector to rotate (query or key)
            float v0 = vec[i];
            float v1 = vec[i+1];
            vec[i]   = v0 * fcr - v1 * fci;
            vec[i+1] = v0 * fci + v1 * fcr;
        }
    }
}
#endif

#ifdef USE_CUDA
__global__ void multi_head_attention_kernel(int pos, int seq_len, float *sq, float *satt, float *sxb, float *key_cache, float *value_cache, int kv_dim, int kv_mul, int head_size, int loff) {
    int h = blockIdx.x;
    // get the query vector for this head
    float* q = sq + h * head_size;
    // attention scores for this head
    float* att = satt + h * seq_len;
    // iterate over all timesteps, including the current one 
    // In CUDA, each thread does a small portion of the calc
    for (int t = threadIdx.x; t <= pos; t += blockDim.x) {
        // get the key vector for this head and at this timestep
        float* k = key_cache + loff + t * kv_dim + (h / kv_mul) * head_size;
        // calculate the attention score as the dot product of q and k
        float score = 0.0f;
        for (int i = 0; i < head_size; i++) {
            score += q[i] * k[i];
        }
        score /= sqrtf(head_size);
        // save the score to the attention buffer
        att[t] = score;
    }
    // above was this threads portion of the iteration.  wait for all threads to finish
    __syncthreads();

    // softmax the scores to get attention weights, from 0..pos inclusively
    softmax_gpu(att, pos + 1);
    __syncthreads();

    // weighted sum of the values, store back into xb
#if 0
    // FIXME something is wrong with this code...
    float* xb = sxb + h * head_size;
    memset(xb, 0, head_size * sizeof(float));
    for (int t = threadIdx.x; t <= pos; t += blockDim.x) {
        // get the value vector for this head and at this timestep
        float* v = value_cache + loff + t * kv_dim + (h / kv_mul) * head_size;
        // get the attention weight for this timestep
        float a = att[t];
        // accumulate the weighted value into xb
        for (int i = 0; i < head_size; i++) {
            xb[i] += a * v[i];
        }
    }
    __syncthreads();  // FIXME necessary or not?
#else
    // llama2.cu reversed the for loops & refactored...
    for (int i = threadIdx.x; i < head_size; i += blockDim.x) {
        float val = 0.0f;
        for (int t = 0; t <= pos; t++) {
            val += att[t] * value_cache[loff + t * kv_dim + (h / kv_mul) * head_size + i];
        }
        sxb[h * head_size + i] = val;
    }
#endif
}
void multi_head_attention(int pos, Config* p, RunState* s, int kv_dim, int kv_mul, int head_size, int loff) {
    multi_head_attention_kernel <<<p->n_heads, 1024>>> (pos, p->seq_len, s->q, s->att, s->xb, s->key_cache, s->value_cache, kv_dim, kv_mul, head_size, loff);
}
#else
void multi_head_attention(int pos, Config* p, RunState* s, int kv_dim, int kv_mul, int head_size, int loff) {
    int h;
    #pragma omp parallel for private(h)
    for (h = 0; h < p->n_heads; h++) {
        // get the query vector for this head
        float* q = s->q + h * head_size;
        // attention scores for this head
        float* att = s->att + h * p->seq_len;
        // iterate over all timesteps, including the current one
        for (int t = 0; t <= pos; t++) {
            // get the key vector for this head and at this timestep
            float* k = s->key_cache + loff + t * kv_dim + (h / kv_mul) * head_size;
            // calculate the attention score as the dot product of q and k
            float score = 0.0f;
            for (int i = 0; i < head_size; i++) {
                score += q[i] * k[i];
            }
            score /= sqrtf(head_size);
            // save the score to the attention buffer
            att[t] = score;
        }

        // softmax the scores to get attention weights, from 0..pos inclusively
        softmax(att, pos + 1);

        // weighted sum of the values, store back into xb
        float* xb = s->xb + h * head_size;
        memset(xb, 0, head_size * sizeof(float));
        for (int t = 0; t <= pos; t++) {
            // get the value vector for this head and at this timestep
            float* v = s->value_cache + loff + t * kv_dim + (h / kv_mul) * head_size;
            // get the attention weight for this timestep
            float a = att[t];
            // accumulate the weighted value into xb
            for (int i = 0; i < head_size; i++) {
                xb[i] += a * v[i];
            }
        }
    }
}
#endif

#ifdef USE_CUDA
__global__ void f_silu_elementwise_mul_w3_kernel(float *shb, float *shb2, int hidden_dim) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < hidden_dim) {
        float val = shb[i] * (1.0f / (1.0f + expf(-shb[i])));
        shb[i] = val * shb2[i];
    }
}
void f_silu_elementwise_mul_w3(RunState *s, int hidden_dim) {
    f_silu_elementwise_mul_w3_kernel<<<divUp(hidden_dim, 256), 256>>>(s->hb, s->hb2, hidden_dim);
}
#else
void f_silu_elementwise_mul_w3(RunState *s, int hidden_dim) {
    for (int i = 0; i < hidden_dim; i++) {
        s->hb[i] = s->hb[i] * (1.0f / (1.0f + expf(-s->hb[i])));
    }

    // elementwise multiply with w3(x)
    for (int i = 0; i < hidden_dim; i++) {
        s->hb[i] = s->hb[i] * s->hb2[i];
    }
}
#endif

#ifdef USE_CUDA
__global__ void accum_kernel(float* a, float* b, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        a[i] += b[i];
    }
}
void accum(float *a, float *b, int size) {
    accum_kernel<<<divUp(size, 256), 256>>>(a,b,size);
}
#else
void accum(float *a, float *b, int size) {
    for (int i = 0; i < size; i++) {
        a[i] += b[i];
    }
}
#endif

void transformer(int token, int pos, Config* p, RunState* s, TransformerWeights* w) {

    // a few convenience variables
    float *x = s->x;
    int dim = p->dim;
    int kv_dim = (p->dim * p->n_kv_heads) / p->n_heads;
    int kv_mul = p->n_heads / p->n_kv_heads; // integer multiplier of the kv sharing in multiquery
    int hidden_dim =  p->hidden_dim;
    int head_size = dim / p->n_heads;

    // copy the token embedding into x
    float* content_row = &(w->token_embedding_table[token * dim]);
#ifdef USE_CUDA
    CUCHK(cudaMemcpy(x, content_row, dim*sizeof(*x), cudaMemcpyHostToDevice));
#else
    memcpy(x, content_row, dim*sizeof(*x));
#endif

    // forward all the layers
    for(int l = 0; l < p->n_layers; l++) {

        // attention rmsnorm
        rmsnorm(s->xb, x, w->rms_att_weight + l*dim, dim);

        // qkv matmuls for this position
        matmul(s->q, s->xb, w->wq + l*dim*dim, dim, dim);
        matmul(s->k, s->xb, w->wk + l*dim*kv_dim, dim, kv_dim);
        matmul(s->v, s->xb, w->wv + l*dim*kv_dim, dim, kv_dim);

        // RoPE relative positional encoding: complex-valued rotate q and k in each head
        RoPe_rotation(pos, s, dim, kv_dim, head_size);

        // save key,value at this time step (pos) to our kv cache
        int loff = l * p->seq_len * kv_dim; // kv cache layer offset for convenience
        float* key_cache_row = s->key_cache + loff + pos * kv_dim;
        float* value_cache_row = s->value_cache + loff + pos * kv_dim;
#ifdef USE_CUDA
        CUCHK(cudaMemcpyAsync(key_cache_row, s->k, kv_dim * sizeof(*key_cache_row), cudaMemcpyDeviceToDevice));
        CUCHK(cudaMemcpyAsync(value_cache_row, s->v, kv_dim * sizeof(*value_cache_row), cudaMemcpyDeviceToDevice));
#else
        memcpy(key_cache_row, s->k, kv_dim * sizeof(*key_cache_row));
        memcpy(value_cache_row, s->v, kv_dim * sizeof(*value_cache_row));
#endif

        // multihead attention. iterate over all heads
        multi_head_attention(pos, p, s, kv_dim, kv_mul, head_size, loff);

        // final matmul to get the output of the attention
        matmul(s->xb2, s->xb, w->wo + l*dim*dim, dim, dim);

        // residual connection back into x
        accum(x, s->xb2, dim);

        // ffn rmsnorm
        rmsnorm(s->xb, x, w->rms_ffn_weight + l*dim, dim);

        // Now for FFN in PyTorch we have: self.w2(F.silu(self.w1(x)) * self.w3(x))
        // first calculate self.w1(x) and self.w3(x)
        matmul(s->hb, s->xb, w->w1 + l*dim*hidden_dim, dim, hidden_dim);
        matmul(s->hb2, s->xb, w->w3 + l*dim*hidden_dim, dim, hidden_dim);

        // F.silu; silu(x)=x*σ(x),where σ(x) is the logistic sigmoid
        // elementwise multiply with w3(x)
        f_silu_elementwise_mul_w3(s, hidden_dim);

        // final matmul to get the output of the ffn
        matmul(s->xb, s->hb, w->w2 + l*dim*hidden_dim, hidden_dim, dim);

        // residual connection
        accum(x, s->xb, dim);
    }

    // final rmsnorm
    rmsnorm(x, x, w->rms_final_weight, dim);

    // classifier into logits
#ifdef USE_CUDA
    matmul(s->logits_gpu, x, w->wcls, p->dim, p->vocab_size);
    CUCHK(cudaMemcpy(s->logits, s->logits_gpu, p->vocab_size * sizeof(float), cudaMemcpyDeviceToHost));
#else
    matmul(s->logits, x, w->wcls, p->dim, p->vocab_size);
#endif 

}

// ----------------------------------------------------------------------------
// The Byte Pair Encoding (BPE) Tokenizer that translates strings <-> tokens

typedef struct {
    char** vocab;
    float* vocab_scores;
    int vocab_size;
    unsigned int max_token_length;
    char byte_piece[2];
} Tokenizer;

void build_tokenizer(char* tokenizer, Tokenizer* t, int vocab_size) {
    // i should have written the vocab_size into the tokenizer file... sigh
    t->vocab_size = vocab_size;
    // malloc space to hold the scores and the strings
    t->vocab = (char**)malloc(vocab_size * sizeof(char*));
    t->vocab_scores = (float*)malloc(vocab_size * sizeof(float));
    t->byte_piece[1] = '\0'; // null terminate the byte_piece string
    // read in the file
    FILE *file = fopen(tokenizer, "rb");
    if (!file) { fprintf(stderr, "couldn't load %s\n", tokenizer); exit(EXIT_FAILURE); }
    if (fread(&t->max_token_length, sizeof(int), 1, file) != 1) { fprintf(stderr, "failed read\n"); exit(EXIT_FAILURE); }
    int len;
    for (int i = 0; i < vocab_size; i++) {
        if (fread(t->vocab_scores + i, sizeof(float), 1, file) != 1) { fprintf(stderr, "failed read\n"); exit(EXIT_FAILURE);}
        if (fread(&len, sizeof(int), 1, file) != 1) { fprintf(stderr, "failed read\n"); exit(EXIT_FAILURE); }
        t->vocab[i] = (char *)malloc(len + 1);
        if (fread(t->vocab[i], len, 1, file) != 1) { fprintf(stderr, "failed read\n"); exit(EXIT_FAILURE); }
        t->vocab[i][len] = '\0'; // add the string terminating token
    }
    fclose(file);
}

void free_tokenizer(Tokenizer* t) {
    for (int i = 0; i < t->vocab_size; i++) {
        free(t->vocab[i]);
    }
    free(t->vocab);
    free(t->vocab_scores);
}

char* get_piece(Tokenizer* t, int prev_token, int token) {
    char *piece = t->vocab[token];
    // following BOS (1) token, sentencepiece decoder strips any leading whitespace (see PR #89)
    if (prev_token == 1 && piece[0] == ' ') { piece++; }
    // careful, some tokens designate raw bytes, and look like e.g. '<0x01>'
    unsigned char byte_val;
    if (sscanf(piece, "<0x%02hhX>", &byte_val) == 1) {
        // ok this token is a raw byte token, careful to only print printable chars or whitespace
        // some of the other bytes can be various control codes, backspace, etc. => skip
        if (isprint(byte_val) || isspace(byte_val)) {
            t->byte_piece[0] = byte_val;
            piece = &t->byte_piece[0];
        }
    }
    return piece;
}

typedef struct {
    const char *str;
    int id;
} TokenIndex;

int compare_tokens(const void *a, const void *b) {
    return strcmp(((TokenIndex*)a)->str, ((TokenIndex*)b)->str);
}

int str_lookup(const char *str, TokenIndex *sorted_vocab, int vocab_size) {
    // efficiently find the perfect match for str in vocab, return its index or -1 if not found
    TokenIndex tok = { .str = str }; // acts as the key to search for
    TokenIndex *res = (TokenIndex *)bsearch(&tok, sorted_vocab, vocab_size, sizeof(TokenIndex), compare_tokens);
    return res != NULL ? res->id : -1;
}

void bpe_encode(Tokenizer* t, char *text, int *tokens, int *n_tokens) {
// encode the string text (input) into an upper-bound preallocated tokens[] array

    // sort vocabulary
    TokenIndex *sorted_vocab = (TokenIndex *)malloc(t->vocab_size * sizeof(TokenIndex));
    for (int i = 0; i < t->vocab_size; i++) {
        sorted_vocab[i].str = t->vocab[i];
        sorted_vocab[i].id = i;
    }
    qsort(sorted_vocab, t->vocab_size, sizeof(TokenIndex), compare_tokens);

    // create a temporary buffer that will store merge candidates of always two consecutive tokens
    char* str_buffer = (char *)malloc((t->max_token_length*2 +1 +2) * sizeof(char)); // *2 for concat, +1 for null terminator +2 for UTF8 (in case max_token_lenght is 1)
    size_t str_len = 0;

    // add_dummy_prefix is true by default
    tokens[0] = str_lookup(" ", sorted_vocab, t->vocab_size);
    *n_tokens = 1; // the number of tokens

    // Okay UTF-8 time. This will get messy. Here is the reference from Wikipedia:
    // Code point ↔ UTF-8 conversion
    // First code point	Last code point	Byte 1	Byte 2	Byte 3	Byte 4
    // U+0000	U+007F	    0xxxxxxx
    // U+0080	U+07FF	    110xxxxx	10xxxxxx
    // U+0800	U+FFFF	    1110xxxx	10xxxxxx	10xxxxxx
    // U+10000	U+10FFFF    11110xxx	10xxxxxx	10xxxxxx	10xxxxxx

    // process the raw (UTF-8) byte sequence of the input string
    for (char *c = text; *c != '\0'; c++) {

        // reset buffer if the current byte is ASCII or a leading byte
        // 0xC0 is 11000000, so (*c & 0xC0) keeps the first 2 bits and zeros the rest
        // 0x80 is 10000000
        // in UTF-8, all continuation bytes start with "10" in first two bits
        // so in English this is: "if this byte is not a continuation byte"
        if ((*c & 0xC0) != 0x80) {
            // this byte must be either a leading byte (11...) or an ASCII char (0x...)
            // => reset our location, as we're starting a new UTF-8 codepoint
            str_len = 0;
        }

        // append the current byte to the buffer
        str_buffer[str_len++] = *c; // ++ is post-increment, incremented after this line
        str_buffer[str_len] = '\0';

        // while the next character is a continuation byte, continue appending
        // but if there are too many of them, just stop to avoid overruning str_buffer size.
        if ((*(c+1) & 0xC0) == 0x80 && str_len < 4) {
            continue;
        }

        // ok c+1 is not a continuation byte, so we've read in a full codepoint
        int id = str_lookup(str_buffer, sorted_vocab, t->vocab_size);

        if (id != -1) {
            // we found this codepoint in vocab, add it as a token
            tokens[(*n_tokens)++] = id;
        } else {
            // byte_fallback encoding: just encode each byte as a token
            // +3 is here because the first 3 vocab elements are <unk>, <s>, </s>
            // so the individual bytes only start at index 3
            for (int i=0; i < str_len; i++) {
                tokens[(*n_tokens)++] = (unsigned char)str_buffer[i] + 3;
            }
        }
        str_len = 0; // protect against a sequence of stray UTF8 continuation bytes
    }

    // merge the best consecutive pair each iteration, according the scores in vocab_scores
    while (1) {
        float best_score = -1e10;
        int best_id = -1;
        int best_idx = -1;

        for (int i=0; i < (*n_tokens-1); i++) {
            // check if we can merge the pair (tokens[i], tokens[i+1])
            sprintf(str_buffer, "%s%s", t->vocab[tokens[i]], t->vocab[tokens[i+1]]);
            int id = str_lookup(str_buffer, sorted_vocab, t->vocab_size);
            if (id != -1 && t->vocab_scores[id] > best_score) {
                // this merge pair exists in vocab! record its score and position
                best_score = t->vocab_scores[id];
                best_id = id;
                best_idx = i;
            }
        }

        if (best_idx == -1) {
            break; // we couldn't find any more pairs to merge, so we're done
        }

        // merge the consecutive pair (best_idx, best_idx+1) into new token best_id
        tokens[best_idx] = best_id;
        // delete token at position best_idx+1, shift the entire sequence back 1
        for (int i = best_idx+1; i < (*n_tokens-1); i++) {
            tokens[i] = tokens[i+1];
        }
        (*n_tokens)--; // token length decreased
    }

    free(str_buffer);
    free(sorted_vocab);
}

// ----------------------------------------------------------------------------
// utilities: time / rng

long time_in_ms() {
    // return time in milliseconds, for benchmarking the model speed
    struct timespec time;
    clock_gettime(CLOCK_REALTIME, &time);
    return time.tv_sec * 1000 + time.tv_nsec / 1000000;
}

unsigned long long rng_seed;
unsigned int random_u32() {
    // xorshift rng: https://en.wikipedia.org/wiki/Xorshift#xorshift.2A
    rng_seed ^= rng_seed >> 12;
    rng_seed ^= rng_seed << 25;
    rng_seed ^= rng_seed >> 27;
    return (rng_seed * 0x2545F4914F6CDD1Dull) >> 32;
}
float random_f32() { // random float32 in [0,1)
    return (random_u32() >> 8) / 16777216.0f;
}

// ----------------------------------------------------------------------------
// sampling can be done in a few ways: greedy argmax, sampling, top-p sampling

typedef struct {
    float prob;
    int index;
} ProbIndex; // struct used when sorting probabilities during top-p sampling

int argmax(float* probabilities, int n) {
    // return the index that has the highest probability
    int max_i = 0;
    float max_p = probabilities[0];
    for (int i = 1; i < n; i++) {
        if (probabilities[i] > max_p) {
            max_i = i;
            max_p = probabilities[i];
        }
    }
    return max_i;
}

int sample(float* probabilities, int n) {
    // sample index from probabilities (they must sum to 1!)
    float r = random_f32();
    float cdf = 0.0f;
    for (int i = 0; i < n; i++) {
        cdf += probabilities[i];
        if (r < cdf) {
            return i;
        }
    }
    return n - 1; // in case of rounding errors
}

int compare(const void* a, const void* b) {
    ProbIndex* a_ = (ProbIndex*) a;
    ProbIndex* b_ = (ProbIndex*) b;
    if (a_->prob > b_->prob) return -1;
    if (a_->prob < b_->prob) return 1;
    return 0;
}

int sample_topp(float* probabilities, int n, float topp, ProbIndex* probindex) {
    // top-p sampling (or "nucleus sampling") samples from the smallest set of
    // tokens that exceed probability topp. This way we never sample tokens that
    // have very low probabilities and are less likely to go "off the rails".

    int n0 = 0;
    // quicksort indices in descending order of probabilities
    // values smaller than (1 - topp) / (n - 1) cannot be part of the result
    // so for efficiency we crop these out as candidates before sorting
    const float cutoff = (1.0f - topp) / (n - 1);
    for (int i = 0; i < n; i++) {
        if (probabilities[i] >= cutoff) {
            probindex[n0].index = i;
            probindex[n0].prob = probabilities[i];
            n0++;
        }
    }
    qsort(probindex, n0, sizeof(ProbIndex), compare);

    // truncate the list where cumulative probability exceeds topp
    float cumulative_prob = 0.0f;
    int last_idx = n0 - 1; // in case of rounding errors consider all elements
    for (int i = 0; i < n0; i++) {
        cumulative_prob += probindex[i].prob;
        if (cumulative_prob > topp) {
            last_idx = i;
            break; // we've exceeded topp by including last_idx
        }
    }

    // sample from the truncated list
    float r = random_f32() * cumulative_prob;
    float cdf = 0.0f;
    for (int i = 0; i <= last_idx; i++) {
        cdf += probindex[i].prob;
        if (r < cdf) {
            return probindex[i].index;
        }
    }
    return probindex[last_idx].index; // in case of rounding errors
}


// ----------------------------------------------------------------------------
// int main

void error_usage() {
    fprintf(stderr, "Usage:   run <checkpoint> [options]\n");
    fprintf(stderr, "Example: run model.bin -n 256 -i \"Once upon a time\"\n");
    fprintf(stderr, "Options:\n");
    fprintf(stderr, "  -t <float>  temperature, default 1.0\n");
    fprintf(stderr, "  -p <float>  p value in top-p (nucleus) sampling. default 0.9\n");
    fprintf(stderr, "  -s <int>    random seed, default time(NULL)\n");
    fprintf(stderr, "  -n <int>    number of steps to run for, default 256. 0 = max_seq_len\n");
    fprintf(stderr, "  -i <string> input prompt\n");
    fprintf(stderr, "  -z <string> optional path to custom tokenizer\n");
    exit(EXIT_FAILURE);
}

int main(int argc, char *argv[]) {

    // default inits
    char *checkpoint_path = NULL;  // e.g. out/model.bin
    char *tokenizer_path = (char *)"tokenizer.bin";
    float temperature = 1.0f; // 0.0 = greedy deterministic. 1.0 = original. don't set higher
    float topp = 0.9f;        // top-p in nucleus sampling. 1.0 = off. 0.9 works well, but slower
    rng_seed = 0; // seed rng with time by default
    int steps = 256;          // number of steps to run for
    char *prompt = NULL;      // prompt string

    // poor man's C argparse so we can override the defaults above from the command line
    if (argc >= 2) { checkpoint_path = argv[1]; } else { error_usage(); }
    for (int i = 2; i < argc; i+=2) {
        // do some basic validation
        if (i + 1 >= argc) { error_usage(); } // must have arg after flag
        if (argv[i][0] != '-') { error_usage(); } // must start with dash
        if (strlen(argv[i]) != 2) { error_usage(); } // must be -x (one dash, one letter)
        // read in the args
        if (argv[i][1] == 't') { temperature = atof(argv[i + 1]); }
        else if (argv[i][1] == 'p') { topp = atof(argv[i + 1]); }
        else if (argv[i][1] == 's') { rng_seed = atoi(argv[i + 1]); }
        else if (argv[i][1] == 'n') { steps = atoi(argv[i + 1]); }
        else if (argv[i][1] == 'i') { prompt = argv[i + 1]; }
        else if (argv[i][1] == 'z') { tokenizer_path = argv[i + 1]; }
        else { error_usage(); }
    }
    if(rng_seed == 0) { rng_seed =  (unsigned int)time(NULL);}

    // read in the model.bin file
    Config config;
    TransformerWeights weights;
    int fd = 0;         // file descriptor for memory mapping
    float* data = NULL; // memory mapped data pointer
    ssize_t file_size;  // size of the checkpoint file in bytes
    read_checkpoint(checkpoint_path, &config, &weights, &fd, &data, &file_size);

    // right now we cannot run for more than config.seq_len steps
    if (steps <= 0 || steps > config.seq_len) { steps = config.seq_len; }

    // read in the tokenizer .bin file
    Tokenizer tokenizer;
    build_tokenizer(tokenizer_path, &tokenizer, config.vocab_size);

    // create and init the application RunState
    RunState state;
    malloc_run_state(&state, &config);
    ProbIndex *probindex = (ProbIndex *)malloc(config.vocab_size * sizeof(ProbIndex)); // buffer used in top-p sampling

    // process the prompt, if any
    int *prompt_tokens = NULL;
    int num_prompt_tokens = 0;
    if (prompt != NULL) {
        prompt_tokens = (int*)malloc((strlen(prompt)+1) * sizeof(int));
        bpe_encode(&tokenizer, prompt, prompt_tokens, &num_prompt_tokens);
    }

    // start the main loop
    long start = 0;  // used to time our code, only initialized after first iteration
    int next;        // will store the next token in the sequence
    int token = 1;   // init with token 1 (=BOS), as done in Llama-2 sentencepiece tokenizer
    int pos = 0;     // position in the sequence
    while (pos < steps) {

        // forward the transformer to get logits for the next token
        transformer(token, pos, &config, &state, &weights);

        // advance the state state machine
        if(pos < num_prompt_tokens) {
            // if we are still processing the input prompt, force the next prompt token
            next = prompt_tokens[pos];
        } else {
            // sample the next token
            if (temperature == 0.0f) {
                // greedy argmax sampling: take the token with the highest probability
                next = argmax(state.logits, config.vocab_size);
            } else {
                // apply the temperature to the logits
                for (int q=0; q<config.vocab_size; q++) { state.logits[q] /= temperature; }
                // apply softmax to the logits to get the probabilities for next token
                softmax(state.logits, config.vocab_size);
                // we sample from this distribution to get the next token
                if (topp <= 0 || topp >= 1) {
                    // simply sample from the predicted probability distribution
                    next = sample(state.logits, config.vocab_size);
                } else {
                    // top-p (nucleus) sampling, clamping the least likely tokens to zero
                    next = sample_topp(state.logits, config.vocab_size, topp, probindex);
                }
            }
        }
        pos++;

        // data-dependent terminating condition: the BOS (1) token delimits sequences
        if (next == 1) { break; }

        // print the token as string, decode it with the Tokenizer object
        char* piece = get_piece(&tokenizer, token, next);
        printf("%s", piece);
        fflush(stdout);
        token = next;

        // init the timer here because the first iteration can be slower
        if (start == 0) { start = time_in_ms(); }
    }
    printf("\n");

    // report achieved tok/s (pos-1 because the timer starts after first iteration)
    if (pos > 1) {
        long end = time_in_ms();
        fprintf(stderr, "achieved tok/s: %f\n", (pos-1) / (double)(end-start)*1000);
    }

    // memory and file handles cleanup
    free_run_state(&state);
    free(probindex);
    free_tokenizer(&tokenizer);
    if (prompt_tokens != NULL) free(prompt_tokens);
    if (data != MAP_FAILED) munmap(data, file_size);
    if (fd != -1) close(fd);
    return 0;
}
